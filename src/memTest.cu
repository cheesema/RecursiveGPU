#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <chrono>
#include <memory>
#include <vector>

template<typename T>
__global__ void memCopy1dKernel(T *in, T *out, size_t len) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        out[idx] = in[idx];
    }
}

template<typename T>
__global__ void memCopy2dA(const T *in, T *out, size_t xLen, size_t yLen) {
    size_t xi = blockIdx.x * blockDim.x + threadIdx.x;
    size_t yi = blockIdx.y * blockDim.y + threadIdx.y;
    if (xi < xLen && yi < yLen) {
        out[yi * xLen + xi] = in[yi * xLen + xi];
    }
}

template<typename T>
__global__ void memCopy2dB(const T *in, T *out, size_t xLen, size_t yLen) {
    size_t xi = blockIdx.x * blockDim.x + threadIdx.x;
    if (xi < xLen) {
        for (size_t idx = xi; idx < yLen * xLen; idx += xLen) {
            __syncthreads(); // don't need sychronization but it gives super speedup!
            out[idx] = in[idx];
        }
    }
}

template<typename T>
__global__ void memCopy2dBnotSynchronized(const T *in, T *out, size_t xLen, size_t yLen) {
    size_t xi = blockIdx.x * blockDim.x + threadIdx.x;
    if (xi < xLen) {
        for (size_t idx = xi; idx < yLen * xLen; idx += xLen) {
            out[idx] = in[idx];
        }
    }
}

static void waitForCuda() {
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
}

class ScopedBandwidth {
    const std::string name;
    const std::chrono::high_resolution_clock::time_point startTime;
    const size_t dataSize;
    const int numOfRepetitions;
    static constexpr double gigabyte = 1000 * 1000 * 1000;

public:
    ScopedBandwidth(const std::string &name, size_t dataSize, size_t numOfRepetitions) : name(name), startTime(std::chrono::high_resolution_clock::now()), dataSize(dataSize), numOfRepetitions(numOfRepetitions) {}
    ~ScopedBandwidth() {
        auto stopTime = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = stopTime - startTime;
        std::cout << name << " GB/s: " << (2 * dataSize * numOfRepetitions) / elapsed.count() / gigabyte << std::endl;
    }
};


template <typename T>
void warmUp(int numOfRepetitions, int numOfThreads, size_t xLen, size_t yLen, T *dInput, T *dOutput) {
    const size_t numOfElements = xLen * yLen;

    dim3 threadsPerBlock(numOfThreads);
    dim3 numBlocks((numOfElements + threadsPerBlock.x - 1) / threadsPerBlock.x);

    {
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy1dKernel << < numBlocks, threadsPerBlock >> > (dInput, dOutput, numOfElements);
        }
        waitForCuda();
    }
}

template <typename T>
void test1D(int numOfRepetitions, int numOfThreads, size_t xLen, size_t yLen, T *dInput, T *dOutput) {
    const size_t numOfElements = xLen * yLen;
    const size_t dataSize = numOfElements * sizeof(T);

    dim3 threadsPerBlock(numOfThreads);
    dim3 numBlocks((numOfElements + threadsPerBlock.x - 1) / threadsPerBlock.x);

    {
        ScopedBandwidth sb("test1D", dataSize, numOfRepetitions);
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy1dKernel << < numBlocks, threadsPerBlock >> > (dInput, dOutput, numOfElements);
        }
        waitForCuda();
    }
}

template <typename T>
void test2DA(int numOfRepetitions, int numOfThreads, size_t xLen, size_t yLen, T *dInput, T *dOutput) {
    const size_t numOfElements = xLen * yLen;
    const size_t dataSize = numOfElements * sizeof(T);

    dim3 threadsPerBlock(numOfThreads, 1);
    dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (yLen + threadsPerBlock.y - 1) / threadsPerBlock.y);

    {
        ScopedBandwidth sb("test2DA", dataSize, numOfRepetitions);
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy2dA <<< numBlocks, threadsPerBlock >>> (dInput, dOutput, xLen, yLen);
        }
        waitForCuda();
    }
}

template <typename T>
void test2DB(int numOfRepetitions, int numOfThreads, size_t xLen, size_t yLen, T *dInput, T *dOutput) {
    const size_t numOfElements = xLen * yLen;
    const size_t dataSize = numOfElements * sizeof(T);

    dim3 threadsPerBlock(numOfThreads);
    dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x);

    {
        ScopedBandwidth sb("test2DB", dataSize, numOfRepetitions);
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy2dB <<< numBlocks, threadsPerBlock >>> (dInput, dOutput, xLen, yLen);
        }
        waitForCuda();
    }
}

template <typename T>
void test2DBnotSynchronized(int numOfRepetitions, int numOfThreads, size_t xLen, size_t yLen, T *dInput, T *dOutput) {
    const size_t numOfElements = xLen * yLen;
    const size_t dataSize = numOfElements * sizeof(T);

    dim3 threadsPerBlock(numOfThreads);
    dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x);

    {
        ScopedBandwidth sb("test2DBnotSynchronized", dataSize, numOfRepetitions);
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy2dBnotSynchronized <<< numBlocks, threadsPerBlock >>> (dInput, dOutput, xLen, yLen);
        }
        waitForCuda();
    }
}

template <typename T>
void setMemory(T *dInput, T *dOutput, size_t dataSize) {
    hipMemset(dInput, 0, dataSize);
    hipMemset(dOutput, 11, dataSize);
}

template <typename T>
void checkMemory(T *dOutput, size_t numOfElements) {
    std::unique_ptr<T[]> hMemory(new T[numOfElements]);
    hipMemcpy(hMemory.get(), dOutput, numOfElements * sizeof(T), hipMemcpyDeviceToHost);
    size_t cnt = 0;
    for (size_t i = 0; i < numOfElements; ++i) {
        if (hMemory[i] != 0) { // check if memory was zeroed
            cnt++;
        }
    }

    if (cnt > 0) {
        std::cout << "Memory not copied properly! Found " << numOfElements << "/" << cnt << " errors." << std::endl;
        exit(1);
    }
}

#include "deviceQuery.cuh"


int main() {

    std::cout << getDeviceInfo() << std::endl;

    int deviceId = -1;
    hipGetDevice(&deviceId);
    std::cout  << "Device ID = " << deviceId << std::endl;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    std::cout << "Shared memory per SM = " << deviceProp.sharedMemPerMultiprocessor << std::endl;
    const int smCount = deviceProp.multiProcessorCount;
    std::cout << "Name [" << deviceProp.name << "]\n";
    std::cout << "Number of SMs = " << smCount << std::endl;
    std::cout << "Size of global mem = " << static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f) << "MB" << std::endl;

    using T=float;
    const size_t yLen = 1024;
    // calculate coefficient to get for both buffers about half of memory in GPU
    const size_t halfMemCoef = deviceProp.totalGlobalMem / (yLen * 32 * 64 * smCount * sizeof(T)) / 2 / 2;
    // max efficiency at #SM * 32 (max num of active blocks) * 64 (maximum num of active threads 2048/32 active blocks)
    const size_t xLen = smCount * 32 * 64 * halfMemCoef;
    const size_t numOfElements = xLen * yLen;
    const size_t dataSize = numOfElements * sizeof(T);
    std::cout << "Used DataSize = " << dataSize / 1024 / 1024 << "MB\n";

    T *dInput;
    hipMalloc(&dInput, dataSize);
    T *dOutput;
    hipMalloc(&dOutput, dataSize);

    const int numOfRepetitions = 50;

    warmUp(numOfRepetitions, 64, xLen, yLen, dInput, dOutput);

    std::vector<int> numThreadsPool = {32, 64, 128, 256};

    for (int numOfThreads : numThreadsPool) {
        std::cout << "-------- numOfThreads in block = " << numOfThreads << std::endl;

        setMemory(dInput, dOutput, dataSize);
        test1D(numOfRepetitions, numOfThreads, xLen, yLen, dInput, dOutput);
        checkMemory(dOutput, numOfElements);

        setMemory(dInput, dOutput, dataSize);
        test2DA(numOfRepetitions, numOfThreads, xLen, yLen, dInput, dOutput);
        checkMemory(dOutput, numOfElements);

        setMemory(dInput, dOutput, dataSize);
        test2DB(numOfRepetitions, numOfThreads, xLen, yLen, dInput, dOutput);
        checkMemory(dOutput, numOfElements);

        setMemory(dInput, dOutput, dataSize);
        test2DBnotSynchronized(numOfRepetitions, numOfThreads, xLen, yLen, dInput, dOutput);
        checkMemory(dOutput, numOfElements);
    }

    hipFree(dInput);
    hipFree(dOutput);

    return 0;
}
