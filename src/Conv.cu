#include "hip/hip_runtime.h"
#include "Conv.h"

#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

#ifndef APR_USE_CUDA
#define APR_USE_CUDA
#endif
#include "misc/CudaTools.hpp"

namespace {
    void foo() {
        MeshData<float> f(0, 0, 0);
        MeshData<uint16_t > u16(0, 0, 0);
        computeConv(f, f, f);
        compute3rdPartyConv(f, f, f);
        compute3rdPartyConv(u16, u16, u16);
    }

    template<typename T>
    __global__ void conv(T *in, T *out, int xLen, int yLen, int zLen, T *kernel, int kernelWidth) {
        // Calculate yi/zi coordinates with a ghost layer
        int yi = (blockIdx.y * blockDim.y + threadIdx.y) - blockIdx.y * 2 - 1;
        int zi = (blockIdx.z * blockDim.z + threadIdx.z) - blockIdx.z * 2 - 1;
        int yio = yi;
        int zio = zi;
        if (yi > yLen || zi > zLen) return;

        const unsigned int active = __activemask();
        const int workerIdx = threadIdx.y;

        // Boundary handling (repeat boundary element)
        bool boundary = (yi < 0 || yi == yLen || zi < 0 || zi == zLen || threadIdx.y == 0 || threadIdx.y == 31 || threadIdx.z == 0 || threadIdx.z == 31);
        if (yi < 0) yi = 0;
        if (yi == yLen) yi = yLen - 1;
        if (zi < 0) zi = 0;
        if (zi == zLen) zi = zLen - 1;

        size_t offset = zi * xLen * yLen + yi;

        for (int x = -1; x <= xLen; ++x) { // with boundaries in x-dir
            //TODO: skip reading if already read data
            T v = in[offset];

            T prevElement = __shfl_sync(active, v, workerIdx + blockDim.y - 1, blockDim.y);
            T nextElement = __shfl_sync(active, v, workerIdx + 1, blockDim.y);

            printf("(%d, %d)(%d, %d, %d)[%d] = %f (%f, %f)\n", yio, zio, threadIdx.x, yi, zi, boundary, v, nextElement, prevElement);

            if (x >= 0 && x < xLen - 1) offset += yLen;
            printf("\n");
        }
    }
}

template <typename T>
void computeConv(const MeshData<T> &in, MeshData<T> &out, const MeshData<T> &kernel) {
    std::cout << "HELLO" << std::endl;

    T *dInput;
    size_t dataSize = in.mesh.size() * sizeof(T);
    hipMalloc(&dInput, dataSize);
    hipMemcpy(dInput, in.mesh.get(), dataSize, hipMemcpyHostToDevice);
    T *dOutput;
    hipMalloc(&dOutput, dataSize);
    T *dKernel;
    size_t kernelSize = kernel.mesh.size() * sizeof(T);
    hipMalloc(&dKernel, kernelSize);
    hipMemcpy(dKernel, kernel.mesh.get(), kernelSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1, 32, 32);
    dim3 numBlocks(1,
                   (in.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   (in.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    conv <<<numBlocks, threadsPerBlock>>> (dInput, dOutput, in.x_num, in.y_num, in.z_num, dKernel, kernel.x_num);
    waitForCuda();

    return;
}
//
//#define Mask_width  5
//#define Mask_radius Mask_width/2
//#define TILE_WIDTH 16
//#define w (TILE_WIDTH + Mask_width - 1)
//#define clamp(x) (min(max((x), 0.0), 1.0))
//// 2D version (?)
//__global__ void convolution(float *I, const float* __restrict__ M, float *P,
//                            int channels, int width, int height) {
//    __shared__ float N_ds[w][w];
//    int k;
//    for (k = 0; k < channels; k++) {
//        // First batch loading
//        int dest = threadIdx.y * TILE_WIDTH + threadIdx.x,
//                destY = dest / w, destX = dest % w,
//                srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius,
//                srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
//                src = (srcY * width + srcX) * channels + k;
//        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
//            N_ds[destY][destX] = I[src];
//        else
//            N_ds[destY][destX] = 0;
//
//        // Second batch loading
//        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
//        destY = dest / w, destX = dest % w;
//        srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
//        srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
//        src = (srcY * width + srcX) * channels + k;
//        if (destY < w) {
//            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
//                N_ds[destY][destX] = I[src];
//            else
//                N_ds[destY][destX] = 0;
//        }
//        __syncthreads();
//
//        float accum = 0;
//        int y, x;
//        for (y = 0; y < Mask_width; y++)
//            for (x = 0; x < Mask_width; x++)
//                accum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
//        y = blockIdx.y * TILE_WIDTH + threadIdx.y;
//        x = blockIdx.x * TILE_WIDTH + threadIdx.x;
//        if (y < height && x < width)
//            P[(y * width + x) * channels + k] = clamp(accum);
//        __syncthreads();
//    }
//}

#define     MASK_WIDTH      3
#define     MASK_RADIUS     MASK_WIDTH / 2
#define     TILE_WIDTH      8
#define         W           (TILE_WIDTH + MASK_WIDTH - 1)

/**
 * GPU 3D Convolution using shared memory
 */
 template <typename ImgType>
__global__ void convolution(ImgType *I, ImgType* M, ImgType *P, int width, int height, int depth)
{
    /***** WRITE TO SHARED MEMORY *****/
    __shared__ ImgType N_ds[W][W][W];

    // First batch loading
    int dest = threadIdx.x + (threadIdx.y * TILE_WIDTH) + (threadIdx.z * TILE_WIDTH * TILE_WIDTH);
    int destTmp = dest;
    int destX = destTmp % W;
    destTmp = destTmp / W;
    int destY = destTmp % W;
    destTmp = destTmp / W;
    int destZ = destTmp;

    int srcZ = destZ + (blockIdx.z * TILE_WIDTH) - MASK_RADIUS;
    int srcY = destY + (blockIdx.y * TILE_WIDTH) - MASK_RADIUS;
    int srcX = destX + (blockIdx.x * TILE_WIDTH) - MASK_RADIUS;
    int src = srcX + (srcY * width) + (srcZ * width * height);

    if(srcZ >= 0 && srcZ < depth && srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
        N_ds[destZ][destY][destX] = I[src];
    else
        N_ds[destZ][destY][destX] = 0;

    // Second batch loading
    dest = threadIdx.x + (threadIdx.y * TILE_WIDTH) + (threadIdx.z * TILE_WIDTH * TILE_WIDTH) + TILE_WIDTH * TILE_WIDTH * TILE_WIDTH;

    destTmp = dest;
    destX = destTmp % W;
    destTmp = destTmp / W;
    destY = destTmp % W;
    destTmp = destTmp / W;
    destZ = destTmp;

    srcZ = destZ + (blockIdx.z * TILE_WIDTH) - MASK_RADIUS;
    srcY = destY + (blockIdx.y * TILE_WIDTH) - MASK_RADIUS;
    srcX = destX + (blockIdx.x * TILE_WIDTH) - MASK_RADIUS;
    src = srcX + (srcY * width) + (srcZ * width * height);

    if(destZ < W)
    {
        if(srcZ >= 0 && srcZ < depth && srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destZ][destY][destX] = I[src];
        else
            N_ds[destZ][destY][destX] = 0;
    }
    __syncthreads();

    /***** Perform Convolution *****/
    ImgType sum = 0;
    int z;
    int y;
    int x;
    for(z = 0; z < MASK_WIDTH; z++)
        for(y = 0; y < MASK_WIDTH; y++)
            for(x = 0; x < MASK_WIDTH; x++)
                sum = sum + N_ds[threadIdx.z + z][threadIdx.y + y][threadIdx.x + x] * M[x + (y * MASK_WIDTH) + (z * MASK_WIDTH * MASK_WIDTH)];
    z = threadIdx.z + (blockIdx.z * TILE_WIDTH);
    y = threadIdx.y + (blockIdx.y * TILE_WIDTH);
    x = threadIdx.x + (blockIdx.x * TILE_WIDTH);
    if(z < depth && y < height && x < width)
        P[x + (y * width) + (z * width * height)] = sum;

    __syncthreads();

}

template <typename T>
void compute3rdPartyConv(const MeshData<T> &in, MeshData<T> &out, const MeshData<T> &kernel) {
    std::cout << "HELLO" << std::endl;

    T *dInput;
    size_t dataSize = in.mesh.size() * sizeof(T);
    hipMalloc(&dInput, dataSize);
    hipMemcpy(dInput, in.mesh.get(), dataSize, hipMemcpyHostToDevice);
    T *dOutput;
    hipMalloc(&dOutput, dataSize);
    T *dKernel;
    size_t kernelSize = kernel.mesh.size() * sizeof(T);
    hipMalloc(&dKernel, kernelSize);
    hipMemcpy(dKernel, kernel.mesh.get(), kernelSize, hipMemcpyHostToDevice);

    float mask[] =
            {
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,

                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,

                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f
            };

    T *deviceMaskData;
    hipMalloc((void **)&deviceMaskData,        MASK_WIDTH  * MASK_WIDTH   * MASK_WIDTH  * sizeof(T));
    hipMemcpy(deviceMaskData,       mask, MASK_WIDTH  * MASK_WIDTH   * MASK_WIDTH  * sizeof(T), hipMemcpyHostToDevice);

//    dim3 threadsPerBlock(1, 32, 32);
//    dim3 numBlocks(1,
//                   (in.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
//                   (in.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
//    conv <<<numBlocks, threadsPerBlock>>> (dInput, dOutput, in.x_num, in.y_num, in.z_num, dKernel, kernel.x_num);
    int image_width = in.y_num;
    int image_height = in.x_num;
    int image_depth = in.z_num;
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((image_width + TILE_WIDTH - 1) / TILE_WIDTH, (image_height + TILE_WIDTH - 1) / TILE_WIDTH, (image_depth + TILE_WIDTH - 1) / TILE_WIDTH);
    APRTimer timer(true);
    timer.start_timer("DEVICE time");
    convolution<<<dimGrid, dimBlock>>>(dInput, deviceMaskData, dOutput, image_width, image_height, image_depth);
    waitForCuda();
    timer.stop_timer();
    hipMemcpy(out.mesh.get(), dOutput, dataSize, hipMemcpyDeviceToHost);

    hipFree(deviceMaskData);

    return;
}

// SOME CODE FROM
// https://stackoverflow.com/questions/22577857/3d-convolution-with-cuda-using-shared-memory
// for comparison

int test() {

    int image_width  = 16;
    int image_height = 16;
    int image_depth  = 5;

    float *deviceInputImageData;
    float *deviceOutputImageData;
    float *deviceMaskData;

    float data[] =
            {
                    1.0f,  1.0f,  1.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    2.0f,  2.0f,  2.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    3.0f,  3.0f,  3.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    4.0f,  4.0f,  4.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    5.0f,  5.0f,  5.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    6.0f,  6.0f,  6.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    7.0f,  7.0f,  7.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    8.0f,  8.0f,  8.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    9.0f,  9.0f,  9.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    10.0f, 10.0f, 10.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    11.0f, 11.0f, 11.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    12.0f, 12.0f, 12.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    13.0f, 13.0f, 13.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    14.0f, 14.0f, 14.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    15.0f, 15.0f, 15.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    16.0f, 16.0f, 16.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,

                    1.0f,  1.0f,  1.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    2.0f,  2.0f,  2.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    3.0f,  3.0f,  3.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    4.0f,  4.0f,  4.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    5.0f,  5.0f,  5.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    6.0f,  6.0f,  6.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    7.0f,  7.0f,  7.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    8.0f,  8.0f,  8.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    9.0f,  9.0f,  9.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    10.0f, 10.0f, 10.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    11.0f, 11.0f, 11.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    12.0f, 12.0f, 12.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    13.0f, 13.0f, 13.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    14.0f, 14.0f, 14.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    15.0f, 15.0f, 15.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    16.0f, 16.0f, 16.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,

                    1.0f,  1.0f,  1.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    2.0f,  2.0f,  2.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    3.0f,  3.0f,  3.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    4.0f,  4.0f,  4.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    5.0f,  5.0f,  5.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    6.0f,  6.0f,  6.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    7.0f,  7.0f,  7.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    8.0f,  8.0f,  8.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    9.0f,  9.0f,  9.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    10.0f, 10.0f, 10.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    11.0f, 11.0f, 11.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    12.0f, 12.0f, 12.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    13.0f, 13.0f, 13.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    14.0f, 14.0f, 14.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    15.0f, 15.0f, 15.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    16.0f, 16.0f, 16.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,

                    1.0f,  1.0f,  1.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    2.0f,  2.0f,  2.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    3.0f,  3.0f,  3.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    4.0f,  4.0f,  4.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    5.0f,  5.0f,  5.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    6.0f,  6.0f,  6.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    7.0f,  7.0f,  7.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    8.0f,  8.0f,  8.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    9.0f,  9.0f,  9.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    10.0f, 10.0f, 10.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    11.0f, 11.0f, 11.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    12.0f, 12.0f, 12.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    13.0f, 13.0f, 13.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    14.0f, 14.0f, 14.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    15.0f, 15.0f, 15.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    16.0f, 16.0f, 16.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,

                    1.0f,  1.0f,  1.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    2.0f,  2.0f,  2.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    3.0f,  3.0f,  3.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    4.0f,  4.0f,  4.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    5.0f,  5.0f,  5.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    6.0f,  6.0f,  6.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    7.0f,  7.0f,  7.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    8.0f,  8.0f,  8.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    9.0f,  9.0f,  9.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    10.0f, 10.0f, 10.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    11.0f, 11.0f, 11.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    12.0f, 12.0f, 12.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    13.0f, 13.0f, 13.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    14.0f, 14.0f, 14.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    15.0f, 15.0f, 15.0f, 1.0f, 3.0f, 1.0f, 5.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
                    16.0f, 16.0f, 16.0f, 2.0f, 1.0f, 4.0f, 1.0f, 6.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f
            };

    float mask[] =
            {
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,

                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,

                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f,
                    1.0f, 1.0f, 1.0f
            };

    // CHECK CHECK CHECK CHECK CHECK
    int shared_memory_size = W * W * W;
    int block_size = TILE_WIDTH * TILE_WIDTH * TILE_WIDTH;
    int max_size = 3 * block_size;
    std::cout << "Block Size: " << block_size << " - Shared Memory Size: " << shared_memory_size << " - Max Size: " << max_size << std::endl;
    std::cout << "SHARED MEMORY SIZE HAS TO BE SMALLER THAN MAX SIZE IN ORDER TO WORK PROPERLY !!!!!!!";

    hipMalloc((void **)&deviceInputImageData,  image_width * image_height * image_depth * sizeof(float));
    hipMalloc((void **)&deviceOutputImageData, image_width * image_height * image_depth * sizeof(float));
    hipMalloc((void **)&deviceMaskData,        MASK_WIDTH  * MASK_WIDTH   * MASK_WIDTH  * sizeof(float));

    hipMemcpy(deviceInputImageData, data, image_width * image_height * image_depth * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,       mask, MASK_WIDTH  * MASK_WIDTH   * MASK_WIDTH  * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((image_width + TILE_WIDTH - 1) / TILE_WIDTH, (image_height + TILE_WIDTH - 1) / TILE_WIDTH, (image_depth + TILE_WIDTH - 1) / TILE_WIDTH);
    convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData, image_width, image_height, image_depth);
    hipDeviceSynchronize();

    hipMemcpy(data, deviceOutputImageData, image_width * image_height * image_depth * sizeof(float), hipMemcpyDeviceToHost);

    // Print data
    for(int i = 0; i < image_width * image_height * image_depth; ++i)
    {
        if((i % image_width) == 0)
            std::cout << std::endl;

        if((i % (image_width * image_height)) == 0)
            std::cout << std::endl;

        std::cout << data[i] << " - ";
    }

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    return 0;
}