#include "hip/hip_runtime.h"
#include "memBenchmarks.h"

#include "misc/CudaTools.hpp"


namespace {
    void foo() {
        MeshData<float> f(0, 0, 0);
        MeshData<uint16_t > u16(0, 0, 0);
        memCopy2D(f, f);
        memCopy1D(f, f);
    }
    template<typename T>
    __global__ void memCopy1dKernel(T *in, T *out, size_t len) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < len) {
            out[idx] = in[idx];
        }
    }

    template<typename T>
    __global__ void memCopy2dKernel(T *in, T *out, int xLen, int yLen) {
        int xi = blockIdx.x * blockDim.x + threadIdx.x;
        int yi = blockIdx.y * blockDim.y + threadIdx.y;
        if (xi < xLen && yi < yLen) {
            out[yi * xLen + xi] = in[yi * xLen + xi];
        }
    }

    template<typename T>
    __global__ void memCopy2dTestKernel(T *in, T *out, int xLen, int yLen) {
        int xi = blockIdx.x * blockDim.x + threadIdx.x;
        if (xi < xLen) {
            size_t idx = xi;
            for (int y = 0; y < yLen; ++y) {
                out[idx] = in[idx];
                idx += xLen;
            }
        }
    }

    void printThroughput(const APRTimer &timer, size_t dataSize, const int numOfRepetitions) {
        double t = timer.timings.back() / numOfRepetitions;
        const size_t gigaByte = 1000 * 1000 * 1000;
        // 2* since once read and once written
        std::cout << "Data throughput: " << (double) 2 * dataSize / t / gigaByte << " GB/s in time " << t << std::endl;
    }
}



template <typename T>
void memCopy1D(const MeshData<T> &in, MeshData<T> &out) {
    APRTimer timer(true);

    T *dInput;
    size_t dataSize = in.mesh.size() * sizeof(T);
    hipMalloc(&dInput, dataSize);
    hipMemcpy(dInput, in.mesh.get(), dataSize, hipMemcpyHostToDevice);

    T *dOutput;
    hipMalloc(&dOutput, dataSize);

    dim3 threadsPerBlock(64);
    dim3 numBlocks((in.mesh.size() + threadsPerBlock.x - 1) / threadsPerBlock.x );
    printCudaDims(threadsPerBlock, numBlocks);

    const int numOfRepetitions = 100;
    timer.start_timer("MEM_COPY");
    for (int i = 0; i < numOfRepetitions; ++i) {
        memCopy1dKernel <<< numBlocks, threadsPerBlock >>> (dInput, dOutput, in.mesh.size());
        waitForCuda();
    }
    timer.stop_timer();

    printThroughput(timer, dataSize, numOfRepetitions);

    getDataFromKernel(out, dataSize, dOutput);

    return;
}

template <typename T>
void memCopy2D(const MeshData<T> &in, MeshData<T> &out) {
    APRTimer timer(true);

    T *dInput;
    size_t dataSize = in.mesh.size() * sizeof(T);
    hipMalloc(&dInput, dataSize);
    hipMemcpy(dInput, in.mesh.get(), dataSize, hipMemcpyHostToDevice);

    T *dOutput;
    hipMalloc(&dOutput, dataSize);

    int xLen = in.y_num;
    int yLen = in.x_num;


    const int numOfRepetitions = 100;
    {
        dim3 threadsPerBlock(64, 1);
        dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (yLen + threadsPerBlock.y - 1) / threadsPerBlock.y);
        printCudaDims(threadsPerBlock, numBlocks);
        timer.start_timer("MEM_COPY");
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy2dKernel << < numBlocks, threadsPerBlock >> > (dInput, dOutput, xLen, yLen);
            waitForCuda();
        }
        timer.stop_timer();
        printThroughput(timer, dataSize, numOfRepetitions);
    }
    {
        dim3 threadsPerBlock(64);
        dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x);
        printCudaDims(threadsPerBlock, numBlocks);
        timer.start_timer("MEM_COPY");
        for (int i = 0; i < numOfRepetitions; ++i) {
            memCopy2dTestKernel << < numBlocks, threadsPerBlock >> > (dInput, dOutput, xLen, yLen);
            waitForCuda();
        }
        timer.stop_timer();
        printThroughput(timer, dataSize, numOfRepetitions);
    }
    getDataFromKernel(out, dataSize, dOutput);

    return;
}


